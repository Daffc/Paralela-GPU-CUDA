#include "hip/hip_runtime.h"
// v0.2 modified by WZ

#define GPU_V 750

#if GPU_V == 480
  #define MP 15// number of mutiprocessors (SMs) in GTX480
  #define GRID1 MP*2// GRID sizefor rgb2uintKernelSHM and rgb2uintKernelSHM kernels
  #define NT1 768// number of threads per block in the //   rgb2uintKernelSHM and rgb2uintKernelSHM kernels//    this is perhaps the best value for GTX480
#elif GPU_V == 680
  #define MP 8// number of mutiprocessors (SMs) in GTX680
  #define GRID1 MP*2// GRID sizefor rgb2uintKernelSHM and rgb2uintKernelSHM kernels
  #define NT1 1024           // number of threads per block in the //   rgb2uintKernelSHM and rgb2uintKernelSHM kernels//    this is perhaps the best value for GTX680
#elif GPU_V == 750
  #define MP 5// number of mutiprocessors (SMs) in GTX750Ti
  #define GRID1 MP*2 // GRID sizefor rgb2uintKernelSHM and rgb2uintKernelSHM kernels
  #define NT1 1024           // number of threads per block in the //   rgb2uintKernelSHM and rgb2uintKernelSHM kernels//    this is perhaps the best value for GTX750Ti
#endif


#include "wb4.h" // use our lib instead (under construction)

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLUR_SIZE 5

__global__ void blurKernelSHM(unsigned char *saida, unsigned char  *entrada, int largura, int altura) {
  int linha = blockIdx.y * blockDim.y +threadIdx.y;
  int coluna = blockIdx.x * blockDim.x + threadIdx.x;

  if(linha < altura && coluna < largura){
    int valor_r = 0,
        valor_g = 0,
        valor_b = 0,
        cont = 0,
        linha_atual,
        coluna_atual;

    for(int blurLinha= -BLUR_SIZE; blurLinha <=  BLUR_SIZE; blurLinha++){
      for(int blurColun= -BLUR_SIZE; blurColun <=  BLUR_SIZE; blurColun++){
        linha_atual = linha + blurLinha;
        coluna_atual = coluna + blurColun;

        if((linha_atual >= 0) && (linha_atual < altura) && (coluna_atual >= 0) && (coluna_atual < largura)){
          valor_r += entrada[((linha_atual * largura + coluna_atual) * 3) + 0];
          valor_g += entrada[((linha_atual * largura + coluna_atual) * 3) + 1];
          valor_b += entrada[((linha_atual * largura + coluna_atual) * 3) + 2];
          cont++;
        }
      }
    }

    saida[((linha * largura + coluna) * 3) + 0] = valor_r / cont;
    saida[((linha * largura + coluna) * 3) + 1] = valor_g / cont;
    saida[((linha * largura + coluna) * 3) + 2] = valor_b / cont;
  }
  
}

__global__ void rgb2uintKernelSHM(unsigned int *saida, unsigned char  *entrada, int tamanho){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < tamanho){
    saida[i] = ((unsigned int)entrada[i * 3] << 16) + ((unsigned int)entrada[(i * 3) + 1] << 8) + (unsigned int)(unsigned int)entrada[(i * 3) + 2];
    printf("%d\t%d\t%d\t = %d\n",entrada[i * 3], entrada[(i * 3) + 1 ], entrada[(i * 3) + 2],saida[i]);
  }  
}

__global__ void uint2rgbKernelSHM(unsigned char *saida, unsigned int  *entrada, int tamanho){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if(i < tamanho){
    saida[i * 3] = entrada[i] >> 16;
    saida[(i * 3) + 1] = entrada[i] << 16 >> 24;
    saida[(i * 3) + 2] = entrada[i] << 24 >> 24;
  }
}

//@@ INSERT CODE HERE
  //@@ INSERIR AQUI o codigo do seu kernel CUDA


int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  unsigned char *hostInputImageData;
  unsigned char *hostOutputImageData;
  unsigned char *deviceInputImageData;
  unsigned int  *intDeviceInputImageData;
  unsigned char *deviceOutputImageData;
  unsigned int  *intDeviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 1);
  printf( "imagem de entrada: %s\n", inputImageFile );

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);

// NOW: input and output images are RGB (3 channel)
  outputImage = wbImage_new(imageWidth, imageHeight, 3);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * sizeof(unsigned char) * 3);
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned char) * 3);
  hipMalloc((void **)&intDeviceInputImageData,
             imageWidth * imageHeight * sizeof(unsigned int));
  hipMalloc((void **)&intDeviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned int));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");

  hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * sizeof(unsigned char) * 3, hipMemcpyHostToDevice);

  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////

  // dim3 DimGrid((imageWidth-1)/32 + 1, ((imageHeight)-1)/32+1, 1);
  // dim3 DimBlock(32, 32, 1);

  wbTime_start(Compute, "Doing the computation on the GPU");

  int tamanho = imageWidth * imageHeight;

  dim3 DimGridTrans((tamanho-1)/32 + 1, 1, 1);
  dim3 DimBlockTrans(32, 1, 1);

  rgb2uintKernelSHM<<<DimGridTrans, DimBlockTrans>>>(intDeviceInputImageData, deviceInputImageData, tamanho);
  hipDeviceSynchronize();

  printf("\n---------------------------------------------------------------------------\n");

  uint2rgbKernelSHM<<<DimGridTrans, DimBlockTrans>>>(deviceOutputImageData, intDeviceInputImageData, tamanho);

  // blurKernelSHM<<<DimGrid,DimBlock>>>(deviceOutputImageData, deviceInputImageData, imageWidth, imageHeight);

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");

  hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth * imageHeight * sizeof(unsigned char) * 3, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);
  // DEBUG: if you want to see your image, 
  //   will generate file bellow in current directory
  wbExport( "blurred.ppm", outputImage );

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
