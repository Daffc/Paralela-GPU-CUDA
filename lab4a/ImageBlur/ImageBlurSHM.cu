#include "hip/hip_runtime.h"
// v0.2 modified by WZ

#include "wb4.h" // use our lib instead (under construction)

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLUR_SIZE 5
#define NTHREADS 32
#define BLOCK_SIZE (NTHREADS - (2 * BLUR_SIZE))   // DIMENSÃO DE PIXELS QUE SOFRERÃO O BLUR.
#define TILE_WIDTH (BLOCK_SIZE + (2 * BLUR_SIZE)) // DIMENSÃO TOTAO DE TILE (PIXEL DE BLUR + BORDA).



//@@ INSERT CODE HERE
  //@@ INSERIR AQUI o codigo do seu kernel CUDA
__global__ void blurKernelSHM(unsigned int *saida, unsigned int  *entrada, int largura, int altura) {

  // ESTA PRIMEIRA ETAPA DO KERNEL TEM A FUNÇÃO DE RECUPERAR TODOS OS PIXELS EXISTENTES NA IMAGEM PARA A SHARED MEMORY.
  
  //ALOCAÇÃO DE MEMÓRIA EM SHARED MEMORI PARA TODOS O BLOCO, INCLUINDO AS BORDAS PARA BLUR.
  __shared__ unsigned int sh_mem[TILE_WIDTH][TILE_WIDTH];

  // ENDEREÇO DE "linha", "coluna" e "edereço_imagem"  COM SHIFT PARA ÁREA DE IMAGEM.
  const int coluna = blockIdx.x * BLOCK_SIZE + threadIdx.x - BLUR_SIZE;       
  const int linha = blockIdx.y * BLOCK_SIZE + threadIdx.y - BLUR_SIZE; 
  const int endereco_imagem = (linha * largura) + coluna;                    

  // ARMAZENA EM SHARED MEMORY APENAS OS PIXELS QUE EXITEM NA IMAGEM DE ENTRADA.
  if((linha >= 0) && (coluna >= 0) && (linha < altura) && (coluna < largura)) {

    sh_mem[threadIdx.y][threadIdx.x] = entrada[endereco_imagem]; 
  }
  
  __syncthreads();

  // ESTA SEGUNRA ETAPA DO KERNEL TEM A VERIFICAR SE O PIXEL DE SHARED MEMORY EM QUESTÃO CORRESPONDE A UM DOS QUE DEVERÃO SOFRER BLUR (DENTRO DAS DIMENSÕES [BLOCK_SIZE X BLOCK_SIZE]) E EFETUAR A OPERAÇÃO DE BLUR.

  // VERIFICA SE THREAD TRATA DE UM QUE CORRESPONDA A PIXEL QUE DEVERÁ SOFRER BLUR (OU SEJA, DENTRO DE [BLOCK_SIZE X BLOCK_SIZE])
  if ((threadIdx.x >= BLUR_SIZE) && (threadIdx.x < (TILE_WIDTH - BLUR_SIZE)) && (threadIdx.y >= BLUR_SIZE) && (threadIdx.y < (TILE_WIDTH - BLUR_SIZE))) {
    // VERIFICA SE PIXEL EM QUESTÃO NÃO ESTRAPOLA DIMENSÕES DA IMAGEM ORIGINAL.
    if((linha < altura) && (coluna < largura)){
      
      unsigned int valor_r = 0, valor_g = 0, valor_b = 0, cont = 0;
      int linha_atual,
          coluna_atual,
          linha_imagem,
          coluna_imagem;
  
      for(int blurLinha= -BLUR_SIZE; blurLinha <=  BLUR_SIZE; blurLinha++){
        for(int blurColun= -BLUR_SIZE; blurColun <=  BLUR_SIZE; blurColun++){
          
          // COORDENADAS DE PIXEL EM SHARED MEMORY.
          linha_atual = threadIdx.y  + blurLinha;
          coluna_atual = threadIdx.x + blurColun;

          //COORDENADA DE PIXEL EM IMAGEM REFERÊNCIA.
          linha_imagem = linha + blurLinha;
          coluna_imagem = coluna + blurColun;

          // VERIFICA SE PIXEL QUE SERÁ ADICIONADO A CONTÁGEM DO BLUR [LINHA_IMAGEM, COLUNA_IMAGEM], CONTIDO NA SHARED_MEMORY, 
          // TAMBÉM É UM PIXEL EXISTENE NA IMAGEM ORIGINAL.
          if((linha_imagem >= 0) && (coluna_imagem >= 0) && (linha_imagem < altura) && (coluna_imagem < largura)){
            valor_r += sh_mem[linha_atual][coluna_atual] >> 16;
            valor_g += sh_mem[linha_atual][coluna_atual] << 16 >> 24;
            valor_b += sh_mem[linha_atual][coluna_atual] << 24 >> 24;

            cont++;
          }
          
        }
      }
      // ARMAZENA VALOR DE BLUR NAS TRÊS DIMENSÕES (R,G,B), EM ENTRADA CORRESPONDENTE A PIXEL.
      saida[endereco_imagem] = ((valor_r / cont) << 16) + ((valor_g / cont) << 8) + (valor_b / cont);  
    }
  }  
}

__global__ void rgb2uintKernelSHM(unsigned int *saida, unsigned char  *entrada, int tamanho){
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  // VERIFICA SE PIXEL "i" NÃO EXTRAPOLA IMAGEM ORIGINAL.
  if(i < tamanho){
    // ACESSA COORDENADAS LINEARES DE PIXEL entrada[i * 3] E ARMAZENA SUAS DIMENSÕES (R, G, B) EM INTEIRO NA MATRIZ saida[i]
    saida[i] = ((unsigned int)entrada[i * 3] << 16) + ((unsigned int)entrada[(i * 3) + 1] << 8) + (unsigned int)(unsigned int)entrada[(i * 3) + 2];
  }  
}

__global__ void uint2rgbKernelSHM(unsigned char *saida, unsigned int  *entrada, int tamanho){
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  // VERIFICA SE PIXEL "i" NÃO EXTRAPOLA IMAGEM ORIGINAL.
  if(i < tamanho){
    // REALIZA OPERAÇÕES DE SHIFT PARA RECUPERAR DIMENSÕES (R, G, B) E COLOCA-LAS EM SUAS DEVIDAS POSIÇÕES EM VETOR saida[i], RESPECTIVAMENTE.
    saida[i * 3] = entrada[i] >> 16;
    saida[(i * 3) + 1] = entrada[i] << 16 >> 24;
    saida[(i * 3) + 2] = entrada[i] << 24 >> 24;
  }
}



int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int tamanho_imagem;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  unsigned char *hostInputImageData;
  unsigned char *hostOutputImageData;
  unsigned char *deviceInputImageData;
  unsigned int  *intDeviceInputImageData;
  unsigned char *deviceOutputImageData;
  unsigned int  *intDeviceOutputImageData;
  

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 1);
  printf( "imagem de entrada: %s\n", inputImageFile );

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  tamanho_imagem = imageWidth * imageHeight;

// NOW: input and output images are RGB (3 channel)
  outputImage = wbImage_new(imageWidth, imageHeight, 3);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * sizeof(unsigned char) * 3);
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned char) * 3);
  hipMalloc((void **)&intDeviceInputImageData,
             imageWidth * imageHeight * sizeof(unsigned int));
  hipMalloc((void **)&intDeviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned int));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");

  hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * sizeof(unsigned char) * 3, hipMemcpyHostToDevice);

  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////



  wbTime_start(Compute, "Doing the computation on the GPU");

  // DEFINIDO DIMENSÕES DE GRID E BLOCK LINEARES PARA KERNELS rgb2uintKernelSHM E uint2rgbKernelSHM.
  dim3 DimGridTrans((tamanho_imagem-1)/NTHREADS + 1, 1, 1);
  dim3 DimBlockTrans(NTHREADS, 1, 1);

  // EFETUANDO TRANSIÇÃO DE CHAR -> INT
  rgb2uintKernelSHM<<<DimGridTrans, DimBlockTrans>>>(intDeviceInputImageData, deviceInputImageData, tamanho_imagem);
  hipDeviceSynchronize();


  // DEFININDO GRID PELA RELAÇÃO ÀS DIMENSÇÕES DA IMAGEM E DA QUANTIDADE DE PIXEL QUE RECEBERÃO O BLUR (BLOCK_SIZE)
  dim3 DimGrid((imageWidth-1)/BLOCK_SIZE + 1, ((imageHeight)-1)/BLOCK_SIZE+1, 1);
  // DEFININDO BLOCO BIDIMENSIONAL PARA COMPORTAR TODAS AS THREADS UTILIZADAS (BLOCK_SIZE + (2 * BLUR_SIZE))
  dim3 DimBlock(NTHREADS, NTHREADS, 1);

  // EVICANDO KERNEL PARA OPERAÇÃO DE BLUR EM IMAGEM.
  blurKernelSHM<<<DimGrid,DimBlock>>>(intDeviceOutputImageData, intDeviceInputImageData,  imageWidth, imageHeight);
  hipDeviceSynchronize();

  // EFETUANDO TRANSIÇÃO DE INT -> CHAR
  uint2rgbKernelSHM<<<DimGridTrans, DimBlockTrans>>>(deviceOutputImageData, intDeviceOutputImageData, tamanho_imagem);
  hipDeviceSynchronize();

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");

  hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth * imageHeight * sizeof(unsigned char) * 3, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);
  // DEBUG: if you want to see your image, 
  //   will generate file bellow in current directory
  wbExport( "blurred.ppm", outputImage );

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
